#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <cmath>
#include <ctime>
#include <stdio.h>
#include <iostream>
#include "../../../Library/ParallelThreadProfiler.cuh"
#pragma once
#include <string>
#include <vector>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>
#include <time.h>
#include <fstream>
#include <math.h>
using namespace std;
using clock_value_t = long long;

#define WARP_SIZE 32


__device__ void sleep(clock_value_t sleep_cycles)
{
	clock_value_t start = clock64();
	clock_value_t cycles_elapsed;
	do { cycles_elapsed = clock64() - start; } while (cycles_elapsed < sleep_cycles);
}

#define sharedmemory_ints 6144
const int SLEEP_TIME = 50000000;
__global__ void single_loop(int* limits)
{
	RegisterTimeMarker(0); 
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int M = limits[threadIdx.x];

	__shared__ int s[sharedmemory_ints];
	for(int i = 0; i < sharedmemory_ints; i++)
		s[i] = i;

	sleep(SLEEP_TIME);

	RegisterTimeMarker(2);
}

int main()
{
	hipError_t cudaStatus;
	ParallelThreadProfiler::InitialiseProfiling();

	cout << endl << "GPU computations started..." << endl;
	srand(time(NULL));
	
	int limits[WARP_SIZE] = {};
	for(int i=0;i<WARP_SIZE;i++)
	{
		limits[i]=WARP_SIZE-i;
	}

	int* dev_limits=NULL;
	hipMalloc((void**)&dev_limits,WARP_SIZE*sizeof(int));
	hipMemcpy(dev_limits,limits,WARP_SIZE*sizeof(int), hipMemcpyHostToDevice);

	checkCudaErrors(hipPeekAtLastError());

	const int blocks = 16;
	const int thread_per_block = 128;

	ParallelThreadProfiler::CreateLabel("start",0);
	ParallelThreadProfiler::CreateLabel("switch",1);
	ParallelThreadProfiler::CreateLabel("end",2);
	ParallelThreadProfiler::InitialiseKernelProfiling("single_loop_kernel",blocks*thread_per_block,2);

	single_loop<<<blocks,thread_per_block>>>(dev_limits);

	ParallelThreadProfiler::SaveResults();

	hipFree(dev_limits);
	cudaStatus = hipDeviceReset();
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr,"hipDeviceReset failed!!!");
		return 1;
	}

	cout << endl <<"End" << endl;
	return 0;
}

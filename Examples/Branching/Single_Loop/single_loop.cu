#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <cmath>
#include <ctime>
#include <stdio.h>
#include <iostream>
#include "../../../Library/ParallelThreadProfiler.cuh"
#pragma once
#include <string>
#include <vector>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>
#include <time.h>
#include <fstream>
#include <math.h>
using namespace std;
using clock_value_t = long long;

#define WARP_SIZE 32



__device__ void sleep(clock_value_t sleep_cycles)
{
	clock_value_t start = clock64();
	clock_value_t cycles_elapsed;
	do { cycles_elapsed = clock64() - start; } while (cycles_elapsed < sleep_cycles);
}

__global__ void single_loop(int* limits)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	RegisterTimeMarker(0); 

	const int SLEEP_TIME = 50000000;
	switch(tid % 32)
	{
		case 0:
			RegisterTimeMarker(1);
			sleep(SLEEP_TIME);
			RegisterTimeMarker(2);
			break;
		case 1:
			RegisterTimeMarker(1);
			sleep(SLEEP_TIME*2);
			RegisterTimeMarker(2);
			break;
		
		.
		.
		.
		
		case 31:
			RegisterTimeMarker(1);
			sleep(SLEEP_TIME*32);
			RegisterTimeMarker(2);
			break;
	}

	RegisterTimeMarker(3);
}

int main()
{
	hipError_t cudaStatus;
	ParallelThreadProfiler::InitialiseProfiling();

	cout << endl << "GPU computations started..." << endl;
	srand(time(NULL));
	
	int limits[WARP_SIZE] = {};
	for(int i=0;i<WARP_SIZE;i++)
	{
		limits[i]=WARP_SIZE-i;
	}

	int* dev_limits=NULL;
	hipMalloc((void**)&dev_limits,WARP_SIZE*sizeof(int));
	hipMemcpy(dev_limits,limits,WARP_SIZE*sizeof(int), hipMemcpyHostToDevice);

	checkCudaErrors(hipPeekAtLastError());

	const int blocks = 2;
	const int thread_per_block = 32;

	ParallelThreadProfiler::CreateLabel("start",0);
	ParallelThreadProfiler::CreateLabel("switch_start",1);
	ParallelThreadProfiler::CreateLabel("switch_end",2);
	ParallelThreadProfiler::CreateLabel("end",3);
	ParallelThreadProfiler::InitialiseKernelProfiling("single_loop_kernel",blocks*thread_per_block,4);

	single_loop<<<blocks,thread_per_block>>>(dev_limits);

	ParallelThreadProfiler::SaveResults();

	hipFree(dev_limits);
	hipDeviceSynchronize();
	cudaStatus = hipDeviceReset();
	if(cudaStatus != hipSuccess)
	{
		fprintf(stderr,"hipDeviceReset failed!!!");
		return 1;
	}

	cout << endl <<"End" << endl;
	return 0;
}

